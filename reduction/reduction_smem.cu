

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ double reduction_step(double *array, int n, int idx) {
    
    double reg = idx < n ? array[idx] : 0; // Make sure all the threads have indeed a value - should result in less error
    for (int dist = 16; dist > 0; dist /= 2)
        reg += __shfl_down_sync(-1, reg, dist);

    return reg;

}

__global__ void reduction_smem(double *a, int n, double *res) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double reg = reduction_step(a, n, idx);

    // Allocate shared memory statically
    __shared__ double smem[32];

    if (threadIdx.x % 32 == 0) smem[threadIdx.x/32] = reg;
    __syncthreads();

    idx = threadIdx.x;
    reg = reduction_step(smem, 32, idx);

    if (threadIdx.x == 0) atomicAdd(res, reg);
}

