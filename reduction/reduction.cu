

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void reduction(double *a, int n, double *res) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        double reg = a[idx];

        for (int dist = 16; dist > 0; dist /= 2)
            reg += __shfl_down_sync(-1, reg, dist);

        if (threadIdx.x % 32 == 0) atomicAdd(res, reg);
    }
}
