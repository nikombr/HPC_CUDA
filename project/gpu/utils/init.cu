

#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <omp.h>

__device__ void initialize(double *** u, double *** uold, double *** f, int N, double start_T) {



}
                                                                                                                                                                                                                                                                                                                                   
void init(double *** u, double *** uold, double *** f, int N, double start_T) {

    // Initialize values
    double delta = 2.0/(N+1);
    double fracdelta = (N+1)/2.0;

    for (int i = 0; i <= N+1; i++) {
        for (int j = 0; j <= N+1; j++) {
            for (int k = 0; k <= N+1; k++) {  
                // Set f to zero everywhere 
                f[i][j][k] = 0;
                // Initialize uold to start_T
                uold[i][j][k] = start_T;
                u[i][j][k] = start_T;
            }
        }
    }

    // Overwrite a specific region
    int ux = floor(0.625*fracdelta), uy = floor(0.5*fracdelta), lz = ceil(1.0/3.0*fracdelta), uz = floor(fracdelta);
    for (int i = 1; i <= ux; i++) {
        for (int j = 1; j <= uy; j++) {
            for (int k = lz; k <= uz; k++) {   
                f[i][j][k] = 200;
            }
        }
    }

    // Set the boundary of uold and u
    for (int i = 0; i < N+2; i++) {
        for (int j = 0; j < N+2; j++) {

            uold[0][j][i] = 20.0;
            uold[N+1][j][i] = 20.0;

            u[0][j][i] = 20.0;
            u[N+1][j][i] = 20.0;

            uold[i][0][j] = 0;
            uold[i][N+1][j] = 20.0;

            u[i][0][j] = 0;
            u[i][N+1][j] = 20.0;

            uold[i][j][0] = 20.0;
            uold[i][j][N+1] = 20.0;

            u[i][j][0] = 20.0;
            u[i][j][N+1] = 20.0;
        }
    }

}