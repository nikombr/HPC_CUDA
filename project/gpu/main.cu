#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils/alloc3d.h"
#include "utils/print.h"
#include "utils/init.h"

#ifdef _NO_REDUCTION
#include "no_reduction/jacobi.h"
#endif

#ifdef _REDUCTION
#include "reduction/jacobi.h"
#endif

#ifdef _REDUCTION_ATOMIC
#include "reduction_atomic/jacobi.h"
#endif


#include <omp.h>

int
main(int argc, char *argv[]) {

    // Command line intpu
    if (argc < 5 || argc > 7) {
        printf("Usage: %s N(int) iter_max(int) tolerance(double) start_T(double) [output_type(0, 3 or 4)]\n",argv[0]);
        return(1);
    }

    int     N;
    int 	iter_max = 1000;
    double	tolerance;
    double	start_T;
    int		output_type = 0;
    char	*output_prefix = "poisson_gpu";
    char    *extra_str = "";
    char    *output_ext    = "";
    char	output_filename[FILENAME_MAX];
    double 	***u_h, ***u_d, ***uold_h, ***uold_d, ***f_h, ***f_d;
    //double *u_h, *u_d, *uold_h, *uold_d, *f_h, *f_d;
    double  *u_log, *uold_log, *f_log;
    int     n = 0;

    // Get the parameters from the command line
    N         = atoi(argv[1]);	// grid size
    iter_max  = atoi(argv[2]);  // max. no. of iterations
    tolerance = atof(argv[3]);  // tolerance
    start_T   = atof(argv[4]);  // start T for all inner grid points
    if (argc >= 6) {
	    output_type = atoi(argv[5]);  // ouput type
    }
    if (argc == 7) {
        if (atoi(argv[6]) == 1) {
	        extra_str = "_reduction";  // ouput type
        } else if (atoi(argv[6]) == 2) {
            extra_str = "_no_reduction";  // ouput type
        } else {
            extra_str = "_reduction_atomic";  // ouput type
        }

    }

    // Allocate memory
    u_h     = host_malloc_3d(N+2, N+2, N+2);
    uold_h  = host_malloc_3d(N+2, N+2, N+2);
    f_h     = host_malloc_3d(N+2, N+2, N+2);
    device_malloc_3d(&u_d,&u_log, N+2, N+2, N+2);
    device_malloc_3d(&uold_d, &uold_log, N+2, N+2, N+2);
    device_malloc_3d(&f_d, &f_log, N+2, N+2, N+2);
    /*hipMalloc(&u_d,(N+2)*(N+2)*(N+2)*sizeof(double));
    hipMalloc(&uold_d,(N+2)*(N+2)*(N+2)*sizeof(double));
    hipMalloc(&f_d,(N+2)*(N+2)*(N+2)*sizeof(double));
    hipHostMalloc(&u_h,(N+2)*(N+2)*(N+2)*sizeof(double));
    hipHostMalloc(&uold_h,(N+2)*(N+2)*(N+2)*sizeof(double));
    hipHostMalloc(&f_h,(N+2)*(N+2)*(N+2)*sizeof(double));*/
    
    // Check allocation
    if (u_h == NULL || uold_h == NULL || f_h == NULL || u_d == NULL || uold_d == NULL || f_d == NULL) {
        perror("allocation failed");
        exit(-1);
    }


    // Initialize start and boundary conditions on host
    init(u_h, uold_h, f_h, N, start_T);

    // Do GPU warm-up
    //jacobi(u_d, uold_d, f_d, N, iter_max, &tolerance, &n);

    double start_transfer = omp_get_wtime();
    // Copy initializd array to devices
    hipMemcpy(uold_log, **uold_h, (N+2) * (N+2) * (N+2) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u_log, **u_h, (N+2) * (N+2) * (N+2) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(f_log, **f_h, (N+2) * (N+2) * (N+2) * sizeof(double), hipMemcpyHostToDevice);

    // Call Jacobi iteration
    double start = omp_get_wtime();
    jacobi(u_d, uold_d, f_d, N, iter_max, &tolerance, &n);
    double stop = omp_get_wtime() - start;
    

    hipMemcpy(**uold_h, uold_log, (N+2) * (N+2) * (N+2) * sizeof(double), hipMemcpyDeviceToHost);
    double stop_transfer = omp_get_wtime() - start_transfer;
    printf("%d %d %.5f %.5f %.5e # N iterations time transfer_time error\n", N, n, stop, stop_transfer, tolerance);

    // Dump  results if wanted 
    switch(output_type) {
	case 0:
	    // No output at all
	    break;
	case 3:
	    output_ext = ".bin";
	    sprintf(output_filename, "results/%s_%d%s%s", output_prefix, N,extra_str, output_ext);
	    fprintf(stderr, "\nWrite binary dump to %s\n", output_filename);
	    print_binary(output_filename, N, uold_h);
	    break;
	case 4:
	    output_ext = ".vtk";
	    sprintf(output_filename, "results/%s_%d%s", output_prefix, N, output_ext);
	    fprintf(stderr, "\nWrite VTK file to %s\n", output_filename);
	    //print_vtk(output_filename, N, u_h);
	    break;
	default:
	    fprintf(stderr, "Non-supported output type!\n");
	    break;
    }

    // De-allocate memory
    host_free_3d(u_h); host_free_3d(f_h); host_free_3d(uold_h);
    device_free_3d(u_d, u_log); device_free_3d(f_d,f_log); device_free_3d(uold_d,uold_log);
    //hipFree(u_d); hipFree(uold_d); hipFree(f_d);
    //hipHostFree(u_h); hipHostFree(uold_h); hipHostFree(f_h);

    return(0);
}
