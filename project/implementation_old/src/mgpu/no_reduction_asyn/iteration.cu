
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, int start, int end, double delta2, double frac) {
    
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + start;
    if ((i < end + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        //u[i][j][k]  =  40;
    }
}

void iteration(double *** u, double *** uold, double *** f, int N, int start, int end) {
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(end-start+1+dimBlock.z-1)/dimBlock.z);
    // Kernel call
    //cudaDeviceSynchronize();
    //double start = omp_get_wtime();
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, start, end, delta2, frac);
    hipDeviceSynchronize();
    //double end = omp_get_wtime() - start;
    //printf("Time = %f\n",end);
}