
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, double *res, double delta2, double frac) {
    double val = 0;
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < N + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        val = u[i][j][k] - uold[i][j][k];
    }
    
    // Index of thread
    int idx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    // Reduction
    double reg = val*val;
    for (int dist = 16; dist > 0; dist /= 2)
        reg += __shfl_down_sync(-1, reg, dist);
    __syncthreads();

    // Allocate shared memory statically
    __shared__ double smem[32];

    // Copy to shared memory
    if (idx % 32 == 0) smem[idx/32] = reg;
    __syncthreads();

    // Add the elements in shared memory together
    reg = idx < 32 ? smem[idx] : 0;
    for (int dist = 16; dist > 0; dist /= 2)
        reg += __shfl_down_sync(-1, reg, dist);

    if (idx == 0) atomicAdd(res, reg);
    
}

__global__ void init_zero(double *res) {
    *res = 0.0;
}

void iteration(double *** u, double *** uold, double *** f, int N, double *sum) {
    init_zero<<<1, 1>>>(sum);
    hipDeviceSynchronize();
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;

    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y,(N+dimBlock.z-1)/dimBlock.z);

    // Do iteration
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, sum, delta2, frac);
    hipDeviceSynchronize();
}