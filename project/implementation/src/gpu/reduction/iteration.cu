
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, int iter_max, double *res) {
    double val = 0;
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < N + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        val = u[i][j][k] - uold[i][j][k];
    }
    
    // Index of thread
    int idx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    // Reduction
    double reg = val*val;
    for (int dist = 16; dist > 0; dist /= 2)
        reg += __shfl_down_sync(-1, reg, dist);
    __syncthreads();

    // Allocate shared memory statically
    __shared__ double smem[32];

    // Copy to shared memory
    if (idx % 32 == 0) smem[idx/32] = reg;
    __syncthreads();

    // Add the elements in shared memory together
    reg = idx < 32 ? smem[idx] : 0;
    for (int dist = 16; dist > 0; dist /= 2)
        reg += __shfl_down_sync(-1, reg, dist);

    if (idx == 0) atomicAdd(res, reg);
    
}

void iteration(double *** u, double *** uold, double *** f, int N, int iter_max, double *sum) {

    // Blocks and threads
    dim3 dimBlock(32,8,4);
    dim3 dimGrid(((N+2)+dimBlock.x-1)/dimBlock.x,((N+2)+dimBlock.y-1)/dimBlock.y,((N+2)+dimBlock.z-1)/dimBlock.z);

    // Do iteration
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, iter_max, sum);
    hipDeviceSynchronize();
}