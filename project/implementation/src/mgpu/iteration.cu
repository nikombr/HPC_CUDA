
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include "../../lib/info_struct.h"
//#include "../../lib/poisson.h"

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, int iter_max, int width) {
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < width + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] + \
                           uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);

    }
}

void iteration(double *** u, double *** uold, double *** f, int N, int iter_max, int width) {
    // Blocks and threads
    dim3 dimBlock(32,8,4);
    dim3 dimGrid(((N+2)+dimBlock.x-1)/dimBlock.x,((N+2)+dimBlock.y-1)/dimBlock.y,((width+2)+dimBlock.z-1)/dimBlock.z);
    //dim3 dimGrid(1,1,1);
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, iter_max, width);
    hipDeviceSynchronize();
}