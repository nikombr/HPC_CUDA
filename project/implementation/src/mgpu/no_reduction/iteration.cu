
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include "../../lib/info_struct.h"
//#include "../../lib/poisson.h"

__global__ void iteration_inner(double *** u, double *** uold, double *** f, int N, int width, double delta2, double frac) {
    
    int k = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int i = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < width + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        //u[i][j][k] = width;
    }
}

void iteration(double *** u, double *** uold, double *** f, int N, int iter_max, int width) {
    int device;
    hipGetDevice(&device);
    //printf("data: %d, %d\n",width,device);
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    // Blocks and threads
    dim3 dimBlock(32,4,2);
    dim3 dimGrid(((N+1)+dimBlock.x-1)/dimBlock.x,((N+1)+dimBlock.y-1)/dimBlock.y,((width+1)+dimBlock.z-1)/dimBlock.z);
    //dim3 dimGrid(1,1,1);
    iteration_inner<<<dimGrid, dimBlock>>>(u, uold, f, N, width, delta2, frac);
    hipDeviceSynchronize();
}