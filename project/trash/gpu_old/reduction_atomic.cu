
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <omp.h>

__global__ void iteration(double *** u, double *** uold, double *** f, int N, int iter_max, double *res) {
    double val = 0;
    double delta = 2.0/(N+1), delta2 = delta*delta, frac = 1.0/6.0;
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int j = threadIdx.y + blockIdx.y * blockDim.y + 1;
    int k = threadIdx.z + blockIdx.z * blockDim.z + 1;
    if ((i < N + 1) && (j < N + 1) && (k < N + 1)) {
        u[i][j][k] = frac*(uold[i-1][j][k] + uold[i+1][j][k] + uold[i][j-1][k] + uold[i][j+1][k] \
                        + uold[i][j][k+1] + uold[i][j][k-1] + delta2*f[i][j][k]);
        val = u[i][j][k] - uold[i][j][k];
    }

    double reg = val*val;
    atomicAdd(res,reg);
    
}


void
jacobi(double *** u, double *** uold, double *** f, int N, int iter_max, double* tolerance, int *n) {
    double  *sum_h, *sum_d;
    
    hipHostMalloc(&sum_h, sizeof(double), hipHostMallocDefault);
    hipMalloc(&sum_d, sizeof(double));
    *sum_h = *tolerance + 1;
    
    // Blocks and threads
    dim3 dimBlock(8,8,8);
    dim3 dimGrid(((N+2)+dimBlock.x-1)/dimBlock.x,((N+2)+dimBlock.y-1)/dimBlock.y,((N+2)+dimBlock.z-1)/dimBlock.z);
   
    //printf("%d, %d, %d\n",dimGrid.x,dimGrid.y,dimGrid.z);
    while (*n < iter_max && *sum_h > *tolerance) {
        *sum_h = 0.0;
        hipMemcpy(sum_d, sum_h, sizeof(double), hipMemcpyHostToDevice);
 
        // Do iteration
        iteration<<<dimGrid, dimBlock>>>(u, uold, f, N, iter_max, sum_d);
        hipDeviceSynchronize();

        hipMemcpy(sum_h, sum_d, sizeof(double), hipMemcpyDeviceToHost);
        //printf("%.5e, %.5e\n",*sum_h,*tolerance);
        // Swap addresses
        double ***tmp;
        tmp = u;
        u = uold;
        uold = tmp;
        // Next iteration
        (*n)++;
    }
    *tolerance = *sum_h;
    return;

}
