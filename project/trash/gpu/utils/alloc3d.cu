
#include <hip/hip_runtime.h>
#include <stdlib.h>

double ***
host_malloc_3d(int m, int n, int k) {

    double ***p;
    double *a;

    if (m <= 0 || n <= 0 || k <= 0)
        return NULL;

    hipHostMalloc(&p, m * sizeof(double **) + m * n * sizeof(double *), hipHostMallocDefault);

    if (p == NULL) {
        return NULL;
    }

    for(int i = 0; i < m; i++) {
        p[i] = (double **) p + m + i * n ;
    }

    hipHostMalloc(&a, m * n * k * sizeof(double), hipHostMallocDefault);

    if (a == NULL) {
        hipHostFree(p);
        return NULL;
    }

    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            p[i][j] = a + (i * n * k) + (j * k);
        }
    }

    return p;
}

void
host_free_3d(double ***p) {
    hipHostFree(p[0][0]);
    hipHostFree(p);
}

__global__ void mallocLoops(double***p, double *a,int m, int n, int k) {
    for(int i = 0; i < m; i++) {
        p[i] = (double **) p + m + i * n;
    }

    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            p[i][j] = a + (i * n * k) + (j * k);
        }
    }
}

void
device_malloc_3d(double ****B,double ** b,int m, int n, int k) {

    double ***p;
    double *a;

    if (m <= 0 || n <= 0 || k <= 0)
        *B = NULL;

    hipMalloc(&p, m * sizeof(double **) + m * n * sizeof(double *));

    if (p == NULL) {
        *B = NULL;
    }

    hipMalloc(&a, m * n * k * sizeof(double));

    if (a == NULL) {
        hipFree(p);
        *B = NULL;
    }

    mallocLoops<<<1,1>>>(p, a, m, n, k);

    *B = p;
    *b = a;
}

void device_free_3d(double ***B,double*b) {
    hipFree(b);
    hipFree(B);
}